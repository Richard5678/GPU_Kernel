
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello(){
    printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

__host__ int main(){
    hello<<<2, 2>>>();
    hipDeviceSynchronize();
}